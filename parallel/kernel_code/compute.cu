#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include "helper.hpp"
#include "structs.hu"

#define TILE_SZ_A 128
#define TILE_SZ_B 16
#define TILE_SZ_RATIO (TILE_SZ_A/TILE_SZ_B)
#define BLOCK_SIZE 512

extern "C"{
    #include "compute.h"
}

__global__ void mysgemm(int m, int n, int k, const double *A, const double *B, double* C) {

  /********************************************************************
  *
  * Compute C = A x B
  *   where A is a (m x k) matrix
  *   where B is a (k x n) matrix
  *   where C is a (m x n) matrix
  *
  * Use register and shared memory tiling and thread coarsening
  *
  * NOTE: A and C are column major, B is row major
  *
  ********************************************************************/

  // Macros for accessing flattened matrices
  #define A(row,col) A[(row) + (col)*m]
  #define B(row,col) B[(row)*n + (col)]
  #define C(row,col) C[(row) + (col)*m]

  __shared__ double Bs[TILE_SZ_RATIO][TILE_SZ_B];

  double out[TILE_SZ_B] = {0.0};
  double Ar = 0.0;

  int tx = threadIdx.x;
  int row = blockIdx.x * TILE_SZ_A + tx;
  int col = blockIdx.y * TILE_SZ_B;

  for(int i = 0; i < k; i += TILE_SZ_RATIO)
  {
    Bs[tx / TILE_SZ_B][tx % TILE_SZ_B] = (i + tx / TILE_SZ_B < k && col + tx % TILE_SZ_B < n) ? B(i + tx / TILE_SZ_B, col + tx % TILE_SZ_B) : 0.0;
    __syncthreads();

    for(int j = 0; j < TILE_SZ_RATIO; j++)
    {
      if(row < m && i + j < k)
      {
        Ar = A(row, i + j);
        for(int l = 0; l < TILE_SZ_B; l++)
        {
          out[l] += Ar * Bs[j][l];
        }
      }
    }
    __syncthreads();
  }

  if(row < m)
  {
    for(int i = 0; i < TILE_SZ_B; i++)
    {
      if(col + i < n)
      {
        C(row, col + i) = out[i];
      }
    }
  }
  // SSL Hint (9/6/21): try using just one register for the tile of A 
  // rather than several--in other words, load one value (per thread) 
  // from A and compute using that value rather than loading all values 
  // before doing the computation.  This approach seems to be slightly 
  // faster than the alternative.
  #undef A
  #undef B
  #undef C
}

__global__ void coo_dense_elem_mul(int N, double * v, int * i, int * j, 
                                   double * vector, int m, int n, 
                                   double * v_out)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < N)
    {
        int i_curr = i[idx];
        int j_curr = j[idx];
        if(i_curr < m && j_curr < n)
        {
            v_out[idx] = v[idx] * vector[i_curr * n + j_curr];
        }
    }
}

extern "C" 
void critical_kernel_wrapper(sparse_rcs * HT, 
                             sparse_rcs * P_HT_rcs, 
                             ensemble * e, 
                             ensemble * eT, 
                             sparse_coo * C) {
    
    double *ev, *eTv, *eeTv, *Cv, *CeeTv;
    int *Ci, *Cj;

    int *rcsCi;

    hipsparseStatus_t status;
    hipsparseHandle_t handle;

    status = hipsparseCreate(&handle);
    double * h_valPHT;
    int *h_rowPHT, *h_colPHT;
    P_HT_rcs = (sparse_rcs*)malloc(sizeof(sparse_rcs));
     h_valPHT = (double*)malloc((200000)* sizeof(double));
     h_rowPHT = (int*)malloc((C->m+1) * sizeof(int));
     h_colPHT = (int*)malloc((200000) * sizeof(int));

    timer_start("Allocating GPU memory.");
    hipMalloc((void**) &ev, sizeof(double) * (e->X->m * e->X->n));
    hipMalloc((void**) &eTv, sizeof(double) * (eT->X->m * eT->X->n));
    hipMalloc((void**) &eeTv, sizeof(double) * (e->X->m * eT->X->n));
    hipMalloc((void**) &Ci, sizeof(int) * C->N);
    hipMalloc((void**) &Cj, sizeof(int) * C->N);
    hipMalloc((void**) &Cv, sizeof(double) * C->N);
    hipMalloc((void**) &CeeTv, sizeof(double) * C->N);
    hipMalloc((void**) &rcsCi, sizeof(int) * (C->N + 1));
    timer_stop();

    timer_start("Copying input memory to the GPU.");
    hipMemcpy(ev, e->X->v_vector, sizeof(double) * (e->X->m * e->X->n), hipMemcpyHostToDevice);
    hipMemcpy(eTv, eT->X->v_vector, sizeof(double) * (e->X->m * e->X->n), hipMemcpyHostToDevice);
    hipMemcpy(Ci, C->i, sizeof(int) * C->N, hipMemcpyHostToDevice);
    hipMemcpy(Cj, C->j, sizeof(int) * C->N, hipMemcpyHostToDevice);
    hipMemcpy(Cv, C->v, sizeof(double) * C->N, hipMemcpyHostToDevice);
    timer_stop();

    dim3 dimBlockSgemm(TILE_SZ_A, 1, 1);
    dim3 dimGridSgemm(ceil(e->X->m * 1.0 / TILE_SZ_A), ceil(eT->X->n * 1.0 / TILE_SZ_B), 1);

    dim3 dimBlockCooDense(BLOCK_SIZE, 1, 1);
    dim3 dimGridCooDense(ceil(C->N / (1.0 * BLOCK_SIZE)), 1, 1);

    timer_start("Performing GPU Critical Step computation");
    //Here we multiply e against eT giving us the values in vector eeTv
    mysgemm<<<dimGridSgemm, dimBlockSgemm>>>(e->X->m, eT->X->n, e->X->n, ev, eTv, eeTv);
    hipDeviceSynchronize();
    //Next we multiply the dense eeTv vector against the coo sparse C data
    coo_dense_elem_mul<<<dimGridCooDense, dimBlockCooDense>>>(C->N, Cv, Ci, Cj,
                                                                eeTv, e->X->m, eT->X->n,
                                                                CeeTv);
    hipDeviceSynchronize();
    //The output coo sparse from the previous step is converted to csr
    //Csr simply has a compressed row vector, but is the same as coo otherwise
        //thus Ci is converted to rcsCi
    status = hipsparseXcoo2csr(handle, Ci, C->N, C->m, rcsCi, HIPSPARSE_INDEX_BASE_ZERO);
    hipDeviceSynchronize();
    //Sparse Matrix Multiplication of 2 csr sparse matrices (rcsCi, Cj, CeeTv) & (HT->r, HT->j, HT->v)

    //initializing parameters for cusparseSpGEMM
    // double              alpha       = 1.0f;
    // double              beta        = 0.0f;
    hipsparseOperation_t opA         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    hipsparseOperation_t opB         = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    // hipDataType      computeType = HIP_R_64F;


    //device pointers for HT and PHT (matB and matC, matA is CeeTv)
    double *valHT;
    int *rowHT, *colHT;  
    
    double *valPHT;
    int *rowPHT, *colPHT;

    int *nnzC;

    //allocate and copy HT to device memory
    hipMalloc((void**)&valHT, sizeof(double) * (HT->N));
    hipMalloc((void**)&rowHT, sizeof(int) * (HT->m +1));
    hipMalloc((void**)&colHT, sizeof(int) * (HT->N));

    hipMemcpy(valHT, HT->v, sizeof(double) * (HT->N), hipMemcpyHostToDevice);
    hipMemcpy(rowHT, HT->r, sizeof(int) * (HT->m+1), hipMemcpyHostToDevice);
    hipMemcpy(colHT, HT->j, sizeof(int) * (HT->N), hipMemcpyHostToDevice);

    hipMalloc((void**)&rowPHT, sizeof(int) * (C->m + 1));

    hipsparseMatDescr_t matA, matB, matC;
    hipsparseCreateMatDescr(&matA);
    hipsparseCreateMatDescr(&matB);
    hipsparseCreateMatDescr(&matC);

    hipsparseSetMatType(matA,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(matA,HIPSPARSE_INDEX_BASE_ZERO);  

    hipsparseSetMatType(matB,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(matB,HIPSPARSE_INDEX_BASE_ZERO); 

    hipsparseSetMatType(matC,HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(matC,HIPSPARSE_INDEX_BASE_ZERO); 

    hipsparseXcsrgemmNnz(handle, opA, opB,
                        C->m, HT->n, C->n,
                        matA, C->N, rcsCi, Cj,
                        matB, HT->N, rowHT, colHT,
                        matC, rowPHT, nnzC);

    hipDeviceSynchronize();

    printf("first step finished\n");

    int Cnnz = *nnzC;

    hipMalloc((void**)&colPHT, sizeof(int)* Cnnz);
    hipMalloc((void**)&valPHT, sizeof(double)* Cnnz);

    hipsparseDcsrgemm(handle, opA, opB, C->m, HT->n, C->n,
        matA, C->N,
        CeeTv, rcsCi, Cj,
        matB, HT->N,
        valHT, rowHT, colHT,
        matC,
        valPHT, rowPHT, colPHT); 

    hipDeviceSynchronize();

    printf("cusparsegemm finished\n");

    //allocate and copy PHT from device to host
//    double * h_valPHT;
//    int *h_rowPHT, *h_colPHT;
//    double * h_valPHT = new double[2];
    // std::vector<float> h_valPHT((*nnzC) * sizeof(float)); 
    // hipMemcpy(h_valPHT.data(), valPHT, (*nnzC) * sizeof(double), hipMemcpyDeviceToHost);

    // h_valPHT = (double*)malloc(Cnnz * sizeof(double));
    // h_rowPHT = (int*)malloc((C->m+1) * sizeof(int));
    // h_colPHT = (int*)malloc(Cnnz * sizeof(int));

     hipMemcpy(h_valPHT, valPHT, (*nnzC) * sizeof(double), hipMemcpyDeviceToHost);
     hipMemcpy(h_rowPHT, rowPHT, (C->m+1) * sizeof(int), hipMemcpyDeviceToHost);
     hipMemcpy(h_colPHT, colPHT, (*nnzC) * sizeof(int), hipMemcpyDeviceToHost);

     printf("matrix copied from device to host\n");

//     P_HT_rcs = (sparse_rcs*)malloc(sizeof(sparse_rcs));
     P_HT_rcs->N = (*nnzC);
     P_HT_rcs->m = C->m;
     P_HT_rcs->n = HT->n;
     P_HT_rcs->v = h_valPHT;
     P_HT_rcs->r = h_rowPHT;
     P_HT_rcs->j = h_colPHT;

     printf("P_HT initialized\n");


    
    
    // //descriptors and pointers for compute
    // hipsparseSpMatDescr_t matA, matB, matC;
    // void* dBuf1 = NULL;
    // void* dBuf2 = NULL;
    // size_t bufSize1 = 0;
    // size_t bufSize2 = 0;

    // //create cusparseCsr matrices
    // hipsparseCreateCsr(&matA, C->m, C->n, C->N,
    //                   rcsCi, Cj, CeeTv,
    //                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
    //                   HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    // hipsparseCreateCsr(&matB, HT->m, HT->n, HT->N,
    //                   rowHT, colHT, valHT,
    //                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
    //                   HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    // hipsparseCreateCsr(&matC, C->m, HT->n, 0,
    //                   NULL, NULL, NULL,
    //                   HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
    //                   HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

    // //computing
    // hipsparseSpGEMMDescr_t spgemmDesc;
    // hipsparseSpGEMM_createDescr(&spgemmDesc);

    // hipsparseSpGEMM_workEstimation(handle, opA, opB, 
    //                                   &alpha, matA, matB, &beta, matC,
    //                                   computeType, HIPSPARSE_SPGEMM_DEFAULT,
    //                                   spgemmDesc, &bufSize1, NULL);
    // hipMalloc((void**) &dBuf1, bufSize1);                            

    // hipsparseSpGEMM_workEstimation(handle, opA, opB,
    //                                   &alpha, matA, matB, &beta, matC,
    //                                   computeType, HIPSPARSE_SPGEMM_DEFAULT,
    //                                   spgemmDesc, &bufSize1, dBuf1);
    
    // hipsparseSpGEMM_compute(handle, opA, opB,
    //                            &alpha, matA, matB, &beta, matC,
    //                            computeType, HIPSPARSE_SPGEMM_DEFAULT,
    //                            spgemmDesc, &bufSize2, NULL);
    // hipMalloc((void**) &dBuf2, bufSize2);

    // hipsparseSpGEMM_compute(handle, opA, opB,
    //                                        &alpha, matA, matB, &beta, matC,
    //                                        computeType, HIPSPARSE_SPGEMM_DEFAULT,
    //                                        spgemmDesc, &bufSize2, dBuf2);

    // //computation done. copy sizes of matrixC (intermediate matrix)
    // int64_t C_num_rows, C_num_cols, C_nnz;
    // hipsparseSpMatGetSize(matC, &C_num_rows, &C_num_cols,
    //                                      &C_nnz);

    // //allocate and copy to the device pointers for matrix C
    // hipMalloc((void**) &colPHT, C_nnz * sizeof(int));
    // hipMalloc((void**) &valPHT, C_nnz * sizeof(double));

    // hipsparseCsrSetPointers(matC, rowPHT, colPHT, valPHT);       

    // hipsparseSpGEMM_copy(handle, opA, opB,
    //                         &alpha, matA, matB, &beta, matC,
    //                         computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc);

    // //destroy descriptors
    // hipsparseSpGEMM_destroyDescr(spgemmDesc);
    // hipsparseDestroySpMat(matA);
    // hipsparseDestroySpMat(matB);
    // hipsparseDestroySpMat(matC);
    // hipsparseDestroy(handle);

    // //allocate and copy PHT from device to host
    // double *h_valPHT;
    // int *h_rowPHT, *h_colPHT;

    // h_valPHT = (double*)malloc(C_nnz * sizeof(double));
    // h_rowPHT = (int*)malloc(C_num_rows * sizeof(int));
    // h_colPHT = (int*)malloc(C_num_cols * sizeof(int));

    // hipMemcpy(h_valPHT, valPHT, C_nnz * sizeof(double), hipMemcpyDeviceToHost);
    // hipMemcpy(h_rowPHT, rowPHT, C_num_rows * sizeof(int), hipMemcpyDeviceToHost);
    // hipMemcpy(h_colPHT, colPHT, C_num_cols * sizeof(int), hipMemcpyDeviceToHost);

    // P_HT_rcs = (sparse_rcs*)malloc(sizeof(sparse_rcs));
    // P_HT_rcs->N = C_nnz;
    // P_HT_rcs->m = C_num_rows;
    // P_HT_rcs->n = C_num_cols;
    // P_HT_rcs->v = h_valPHT;
    // P_HT_rcs->r = h_rowPHT;
    // P_HT_rcs->j = h_colPHT;

    timer_stop();

    timer_start("Copying output memory to the CPU");
    timer_stop();

    hipFree(ev);
    hipFree(eTv);
    hipFree(eeTv);
    hipFree(CeeTv);

    fprintf(stderr, "KERNEL RAN SUCCESFULLY\n");
}

// //Create space for sparse rcs on device and return pointer
// sparse_rcs * cudaMalloc_sparse_rcs(int N, int m, int n) {
//   sparse_rcs *A;

//   //arg bound check
//   assert(m >= 0);
//   assert(n >= 0);
//   assert(N >= 0);

//   int dim[3] = {N, m, n};
//   fprintf(stderr, "here0\n");
//   //create struct itself
//   hipMalloc((void**) &A, sizeof(sparse_rcs));
//   fprintf(stderr, "here1\n");

//   //copy dimensions into struct
//   hipMemcpy(A, dim, sizeof(int) * 3, hipMemcpyHostToDevice);
//   fprintf(stderr, "here2\n");

//   //copy all associated data
//   if (m > 0) {
//     int * Am;
//     fprintf(stderr, "here5\n");
//     hipMalloc((void **) &(A->r), sizeof(int) * (m+1));
//     fprintf(stderr, "here6\n");
//   }
//   else {
//     //A->r = NULL;
//   }
//   fprintf(stderr, "here4\n");

//   if (N == 0) {
//     //A->v = NULL;
//     //A->j = NULL;
//   }
//   else {
//     hipMalloc((void **) &(A->v), sizeof(double) * N);
//     //hipMemcpy(A->v, B->v, sizeof(double) * N, hipMemcpyHostToDevice);

//     hipMalloc((void **) &(A->j), sizeof(double) * N);
//     //hipMemcpy(A->j, B->j, sizeof(double) * N, hipMemcpyHostToDevice);
//   }
//   fprintf(stderr, "here3\n");

//   return A;
// }

// //copy data from host sparse rcs to device rcs if dims match
// int cudaMemcpy_sparse_rcs(sparse_rcs * dev, sparse_rcs * host, int hostToDevice)
// {
//     if(hostToDevice == 1)
//     {
//         hipMemcpy(dev->r, host->r, sizeof(int) * (host->m+1), hipMemcpyHostToDevice);
//         hipMemcpy(dev->j, host->j, sizeof(double) * host->N, hipMemcpyHostToDevice);
//         hipMemcpy(dev->v, host->v, sizeof(double) * host->N, hipMemcpyHostToDevice);
//         return 0;
//     }
//     else if(hostToDevice == 0)
//     {
//         hipMemcpy(host->r, dev->r, sizeof(int) * (host->m+1), hipMemcpyDeviceToHost);
//         hipMemcpy(host->j, dev->j, sizeof(double) * host->N, hipMemcpyDeviceToHost);
//         hipMemcpy(host->v, dev->v, sizeof(double) * host->N, hipMemcpyDeviceToHost);
//         return 0;
//     }
//     return -1;
// }

// int cudaFree_sparse_rcs(sparse_rcs * dev)
// {
//     hipFree(dev->r);
//     hipFree(dev->j);
//     hipFree(dev->v);
//     hipFree(dev);
//     return 0;
// }

// full_r * cudaMalloc_full_r(int m, int n)
// {
//     full_r * A;

//     //arg bound check
//     assert(m >= 0);
//     assert(n >= 0);

//     int dims[2] = {m, n};

//     hipMalloc((void**) &(A), sizeof(full_r));
//     hipMemcpy(A, dims, sizeof(int) * 2, hipMemcpyHostToDevice);

//     hipMalloc((void**) &(A->v_vector), m * n * sizeof(double));

//     return A;
// }

// int cudaMemcpy_full_r(full_r * dev, full_r * host, int hostToDevice)
// {
//     if(hostToDevice == 1)
//     {
//         hipMemcpy(dev->v_vector, host->v_vector, host->m * host->n * sizeof(double), hipMemcpyHostToDevice);
//         return 0;
//     }
//     else if(hostToDevice == 0)
//     {
//         hipMemcpy(host->v_vector, dev->v_vector, host->m * host->n * sizeof(double), hipMemcpyDeviceToHost);
//         return 0;
//     }
//     return -1;
// }

// int cudaFree_full_r(full_r * dev)
// {
//     hipFree(dev->v_vector);
//     hipFree(dev);
//     return 0;
// }

// sparse_coo * cudaMalloc_sparse_coo(int m, int n, int N)
// {
//     sparse_coo * A;

//     assert(m >= 0);
//     assert(n >= 0);
//     assert(N >= 0);

//     int dims[3] = {m, n, N};

//     hipMalloc((void**) &A, sizeof(sparse_coo));
//     hipMemcpy(A, dims, 3 * sizeof(int), hipMemcpyHostToDevice);

//     hipMalloc((void**) &(A->v), sizeof(double) * N);
//     hipMalloc((void**) &(A->i), sizeof(int) * N);
//     hipMalloc((void**) &(A->j), sizeof(int) * N);

//     return A;
// }

// int cudaMemcpy_sparse_coo(sparse_coo * dev, sparse_coo * host, int hostToDevice)
// {
//     if(hostToDevice == 1)
//     {
//         hipMemcpy(dev->v, host->v, dev->N * sizeof(double), hipMemcpyHostToDevice);
//         hipMemcpy(dev->i, host->i, dev->N * sizeof(int), hipMemcpyHostToDevice);
//         hipMemcpy(dev->j, host->j, dev->N * sizeof(int), hipMemcpyHostToDevice);
//         return 0;
//     }
//     else if(hostToDevice == 0)
//     {
//         hipMemcpy(host->v, dev->v, dev->N * sizeof(double), hipMemcpyDeviceToHost);
//         hipMemcpy(host->i, dev->i, dev->N * sizeof(int), hipMemcpyDeviceToHost);
//         hipMemcpy(host->j, dev->j, dev->N * sizeof(int), hipMemcpyDeviceToHost);
//         return 0;
//     }
//     return -1;
// }

// int cudaFree_sparse_coo(sparse_coo * dev)
// {
//     hipFree(dev->v);
//     hipFree(dev->j);
//     hipFree(dev->i);
//     hipFree(dev);
//     return 0;
// }
